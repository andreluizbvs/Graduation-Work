#include "hip/hip_runtime.h"
#include "SystemResolution.h"

__global__ void conf_dp_kernel(double* dp, double* dpArray, int size, int *bcon, int *contB)
{
	int desloc = 0;
	int nbdof = size;
	for (int a = 0; a < size; a++)
	{
		if (a - desloc < contB[0])
		{
			if (bcon[a] == 0)
			{
				if (dpArray[a - desloc]>0)
					dp[a] = dpArray[a - desloc];
				else
					dp[a] = 0;
			}
			else
			{
				dp[a] = 0;
				desloc++;
			}
		}
		else
			dp[a] = 0;
	}
}

__global__ void set_bcon_kernel(int offset, Particle3D* particles, double n0p, double dirichlet, int nump, int *bcon)
{
	int a = offset + (blockDim.x * blockIdx.x + threadIdx.x);

	if (particles[a].is_wall()){
		bcon[a] = -1;
	}
	else{
		if ((particles[a].get_n() / n0p) < dirichlet){
			bcon[a] = 1;
		}
		else{
			bcon[a] = 0;
		}
	}
}

__global__ void cal_B_HS_ECS_kernel(int offset, Particle3D* particles, double n0p, double *dt, int *nei, double radius, int nump, int *bcon, int *contTemp, double *temp_b)
{
	unsigned int i = offset + (blockDim.x * blockIdx.x + threadIdx.x);
	int j, l;
	double vx, vy, vz, sum = 0.0;
	double dx = 0.0, dy = 0.0, dz = 0.0, dist = 0.0;
	Point3D Part_j, Part_i, Part_jv, Part_iv;
	double ECS = 0.0;
	double alpha = 0.0;
	double beta = 0.0;

	if (i > 0){
		alpha = ((particles)[i].get_n() - n0p) / n0p; // (n[i]-n0p)/n0p
		beta = (dt[0] / n0p)*(temp_b[i - 1] / (-1.0 / (n0p*dt[0]))); // (dt/n0)*(Dn/Dt)
		ECS = fabs(alpha)*(beta / dt[0]) + fabs(beta)*(alpha / dt[0]);
	}
	sum = 0.0;

	for (l = 1; l <= nei[(i*nump) + 0]; l++){
		j = nei[(i*nump) + l];

		if ((particles)[j].is_wall()){
			continue;
		}
		Part_j = (particles)[i].get_po();
		Part_i = (particles)[j].get_po();

		dx = ((Part_i.x) - (Part_j.x));
		dy = ((Part_i.y) - (Part_j.y));
		dz = ((Part_i.z) - (Part_j.z));

		dist = sqrt(dx*dx + dy*dy + dz*dz);

		Part_jv = (particles)[i].get_v();
		Part_iv = (particles)[j].get_v();

		vx = ((Part_iv.x) - (Part_jv.x));
		vy = ((Part_iv.y) - (Part_jv.y));
		vz = ((Part_iv.z) - (Part_jv.z));

		sum += (radius / (dist*dist*dist))*(dx*vx + dy*vy + dz*vz); /////////////// MPS-HS
	}

	temp_b[i] = (-1.25 / (n0p*dt[0]))*sum + ECS; ///////////// MPS-ECS
	if (!(particles)[i].is_wall()){
		atomicAdd(&contTemp[0], 1);
	}


}

__global__ void cal_B_kernel(int offset, Particle3D* particles, double n0p, double *dt, int *contTemp, double *temp_b)
{
	unsigned int i = offset + (blockDim.x * blockIdx.x + threadIdx.x);

	if (!(particles)[i].is_wall()){
		temp_b[i] = 1.0 / dt[0] / dt[0] * ((particles)[i].get_n() - n0p) / n0p;
		atomicAdd(&contTemp[0], 1);
	}

}

__global__ void set_B_HS_ECS_kernel(int offset, int *bcon, int *contB, double *srcB, double *temp_b, int *contTemp){
	
	//unsigned int a = offset + (blockDim.x * blockIdx.x + threadIdx.x);
	for (int a = 0; a < contTemp[0]; a++){
		if (bcon[a] == 0){
			srcB[contB[0]] = temp_b[a];
			atomicAdd(&contB[0], 1);
		}
	}
}

__global__ void cal_A_HL_kernel(int offset, Particle3D* particles, int *neiICCG, double n0pICCG, double radius_ICCG, double rho, int num_d, int nump, int *bcon, double *temp_a)
{
	unsigned int i = offset + (blockDim.x * blockIdx.x + threadIdx.x);
	int j;
	Point3D Part_j, Part_i;
	double dx = 0.0, dy = 0.0, dz = 0.0, dist = 0.0, val = 1.0;

	for (int a = 1; a <= neiICCG[(i*nump) + 0]; a++)
	{
		j = neiICCG[(i*nump) + a];

		if (bcon[j] == -1){
			temp_a[nump*i + j] = 0.0;
		}
		else{
			dx = 0.0;
			dy = 0.0;
			dz = 0.0;
			dist = 0.0;
			val = 1.0;

			Part_j = (particles)[j].get_po();
			Part_i = (particles)[i].get_po();

			dx = Part_j.x - Part_i.x;
			dy = Part_j.y - Part_i.y;
			dz = Part_j.z - Part_i.z;

			dist = sqrt(dx*dx + dy*dy + dz*dz);

			if (num_d == 2){
				val = 3.0 * radius_ICCG;
			}
			else{
				val = 2.0 * radius_ICCG;
			}
			val = val / (dist*dist*dist);
			val = val / n0pICCG;
			val = val / rho;

			temp_a[nump*i + j] = -val;
			temp_a[nump*i + i] += val;
		}
	}
}

__global__ void set_A_HL_kernel1(int offset, int *bcon, double *temp_a2, double *temp_a, int nump, int *bcon_cont){

	unsigned int a = offset + (blockDim.x * blockIdx.x + threadIdx.x);
	int contRows = 0, contCols = 0;

	//for (int a = 0; a < nump; a++){
		//if (bcon[a] != 0){continue;}
	if (bcon[a] != 0){}else{
		for (int b = 0; b < nump; b++){
			if (bcon[b] != 0){}else{
				//temp_a[nump*contRows + contCols] = temp_a[a*nump + b];
				temp_a2[nump*(a - bcon_cont[a]) + (b - bcon_cont[b])] = temp_a[a*nump + b];
				//contCols++;
			}
		}
		//contRows++;
		//contCols = 0;
		//}
	}
}

__global__ void set_A_HL_kernel2(int offsetX, int offsetY, double *srcA, int *contB, double *temp_a, int nump){
	
	//unsigned int g = offset + (blockDim.x * blockIdx.x + threadIdx.x);
	unsigned int g = offsetX*1024 + blockIdx.x;
	//unsigned int h = offset + (blockDim.y * blockIdx.y + threadIdx.y);
	unsigned int h = offsetY*1024 + threadIdx.x;
	int idxOrig = (nump * g) + h;
	int idxDest = (contB[0] * g) + h;

	srcA[idxDest] = temp_a[idxOrig];
	
}

void SystemResolution::set_A_HL(Particle2D* particles, int *neiICCG, double n0pICCG, double radius_ICCG, double lambda, double rho, double dt, int num_d, int nump, int *bcon)
{
	//vector<double> temp(nump);
	double *temp = new double[nump];
	memset(temp, 0, sizeof(double)*nump);
	//vector<vector<double>> temp_a;
	double *temp_a = new double[nump*nump];
	memset(temp_a, 0, sizeof(double)*nump*nump);
	double *srcA = new double[nump*nump];
	memset(srcA, 0, sizeof(double)*nump*nump);

	int cont_line = 0;
	int j;
	Point2D Part_j, Part_i;
	double dx = 0.0, dy = 0.0, dist = 0.0, val = 1.0;

	for (int i = 0; i < nump; i++)
	{
		for (int a = 1; a <= neiICCG[(i*nump) + 0]; a++)
		{
			j = neiICCG[(i*nump) + a];


			if (bcon[j] == -1)
			{
				temp[j] = 0.0;
			}
			else
			{
				dx = 0.0;
				dy = 0.0;
				dist = 0.0;
				//kernel = 0.0;
				val = 1.0;

				Part_j = (particles)[j].get_po();
				Part_i = (particles)[i].get_po();

				dx = Part_j.x - Part_i.x;
				dy = Part_j.y - Part_i.y;

				dist = sqrt(dx*dx + dy*dy);
				
				if (num_d == 2){
					val = 3.0 * radius_ICCG;
				}
				else{
					val = 2.0 * radius_ICCG;
				}
				val = val / (dist*dist*dist);
				val = val / n0pICCG;
				val = val / rho;

				temp[j] = -val;
				temp[cont_line] += val;
			}
		}

		for (int g = 0; g < nump; g++){
			temp_a[nump*i + g] = temp[g];
		}

		//temp_a.push_back(temp);
		cont_line++;

		/*for (int v = 0; v < nump; v++)
			temp[v] = 0;*/
		memset(temp, 0, sizeof(double)*nump);
	}

	//temp.clear();
	//for (int v = 0; v < nump; v++)
	//	temp[v] = 0;
	memset(temp, 0, sizeof(double)*nump);

	int contRows = 0, contCols = 0;

	for (int a = 0; a < nump; a++){
		if (bcon[a] == 0){
			for (int b = 0; b < nump; b++){
				if (bcon[b] == 0){
					srcA[nump*contRows + contCols] = temp_a[a*nump + b];
					contCols++;
				}
			}
			contRows++;
			contCols = 0;
		}		
	}

	delete [] temp;
	delete [] temp_a;
	//temp.clear();
	//temp_a.clear();
}
//ALBVS end
//
//__host__ __device__ void SystemResolution::system_resolution(double dt, int MaxIterPress, int MinIterPress, double Convergence, int allsize){
//
//	//int nbdof = sizeof(B) / sizeof(B[0]);
//	//printf("a: %d, b: %d\n", sizeof(B) , sizeof(B[0]));
//	size_t nbdof = B.size();
//
//	std::vector<double> Xgmm(nbdof), Bgmm(nbdof);
//
//	gmm::row_matrix< gmm::rsvector<double> > Mgmm(nbdof, nbdof);
//
//	Bgmm = B;
//	//Mgmm = A;
//
//	for (int a = 0; a < B.size(); a++)
//	{
//		//Bgmm[a] = B[a];
//
//		for (int b = 0; b < B.size(); b++)
//		{
//			Mgmm[a][b] = A[a][b];
//			//printf("a: %d, b: %d\n", a, b);
//		}
//	}
//
//	gmm::ilut_precond< gmm::row_matrix< gmm::rsvector<double> > > P(Mgmm, 10, 1e-6);
//
//	gmm::iteration iter(1E-10);
//
//	gmm::csc_matrix<double> M2;
//
//	gmm::clean(Mgmm, 1E-16);
//
//	//gmm::copy(Mgmm, M2);
//	printf("resolvendo\n");
//	gmm::bicgstab(Mgmm, Xgmm, Bgmm, P, iter);
//
//
//	//FILE* debug_;
//	//fopen_s(&debug_, "p_puro.vems", "w");
//
//	//for (int a = 0; a < Xgmm.size(); a++)
//	//{
//	//	fprintf(debug_, "%lf\n", Xgmm[a]);
//	//}
//	//fclose(debug_);
//	//system("pause");
//
//	double *dpArray = (double*)calloc(B.size(), sizeof(double));
//
//	for (int a = 0; a < B.size(); a++)
//		dpArray[a] = Xgmm[a];
//
//	conf_dp(dpArray, allsize);
//	free(dpArray);
//
//	Xgmm.clear();
//	Bgmm.clear();
//
//	Mgmm.clear_mat();
//}
//
//
//__host__ __device__ void SystemResolution::conf_dp(double* dpArray, int size)
//{
//	int desloc = 0;
//	//int nbdof = sizeof(B) / sizeof(B[0]);
//	int nbdof = size;
//	for (int a = 0; a < size; a++)
//	{
//		if (a - desloc < B.size())
//		{
//			if (Bcon[a] == 0)
//			{
//				if (dpArray[a - desloc]>0)
//					dp[a] = dpArray[a - desloc];
//				else
//					dp[a] = 0;
//			}
//			else
//			{
//				dp[a] = 0;
//				desloc++;
//			}
//		}
//		else
//			dp[a] = 0;
//	}
//}
//
__host__ __device__ double* SystemResolution::get_dp()
{
	return this->dp;
}
//
//__host__ __device__ int* SystemResolution::GetBcon()
//{
//	return Bcon;
//}
//
