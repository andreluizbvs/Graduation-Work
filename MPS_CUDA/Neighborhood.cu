#include "hip/hip_runtime.h"
#include "Neighborhood.h"

// C�lculo dos vizinhos: A estrutura usada para armazenar � matriz simples.
// A organiza��o na matriz �: Primeiro elemento da linha "i" indica o n�mero de vizinhos da part�cula "i".
// O resto da linha "i" cont�m o �ndice do vizinho da part�cula "i"
// [n_vizinhos,v1,v2,v3,v4,v5,...,v8]

__global__ void set_nei_kernel(int offset, Particle3D* particles, double r2, int nump, int *nei)
{
	unsigned int i = offset + (blockDim.x * blockIdx.x + threadIdx.x);
	//unsigned int i = offset + (/*blockDim.x **/ blockIdx.x /*+ threadIdx.x*/);
	//unsigned int j = offset + (/*blockDim.x * blockIdx.x +*/ threadIdx.x);

	Point3D Part_j, Part_i;
	double x, y, z;

	for (int j = 0; j < nump; j++){
		if (!((particles[j].is_wall() && particles[i].is_wall()) || (i == j))){
			Part_j = (particles)[j].get_po();
			Part_i = (particles)[i].get_po();

			x = Part_j.x - Part_i.x;
			y = Part_j.y - Part_i.y;
			z = Part_j.z - Part_i.z;

			if (((x*x) + (y*y) + (z*z)) <= r2)
			{
				nei[(i * nump) + 0]++;
				nei[(i * nump) + nei[(i * nump) + 0]] = j;
			}
		}
	}
}

__host__ __device__ neighbor::neighbor(const neighbor &n)
{
	this->neighbors = n.neighbors;
}

__host__ __device__ neighbor::neighbor(int cont, int max_nei)
{
	neighbors = new int*[cont];
	for (int i = 0; i < cont; ++i) neighbors[i] = new int[max_nei];
	for (int i = 0; i < cont; i++) {
		memset(neighbors[i], 0, sizeof(neighbors[i][0]) * max_nei);
	}
}

__host__ __device__ neighbor::~neighbor()
{

}

__host__ __device__ void neighbor::clear(int nump)
{
	for (int i = 0; i < nump; i++) {
		memset(neighbors[i], 0, sizeof(neighbors[i][0]) * nump);
	}
}

__host__ __device__ int** neighbor::get()
{
	return neighbors;
}

__host__ __device__ void neighbor::set(Particle3D* particles, double r2, int nump)
{
	Point3D Part_j, Part_i;
	double x, y, z;

	for (int i = 0; i<nump; i++)
	{
		for (int j = i + 1; j<nump; j++)
		{
			if ((particles)[j].is_wall() && (particles)[i].is_wall())
				continue;

			Part_j = (particles)[j].get_po();
			Part_i = (particles)[i].get_po();

			x = Part_j.x - Part_i.x;
			y = Part_j.y - Part_i.y;
			z = Part_j.z - Part_i.z;

			if (((x*x) + (y*y) + (z*z)) <= r2)
			{
				neighbors[i][0]++;
				neighbors[j][0]++;

				neighbors[i][neighbors[i][0]] = j;
				neighbors[j][neighbors[j][0]] = i;
			}
		}
	}
}
