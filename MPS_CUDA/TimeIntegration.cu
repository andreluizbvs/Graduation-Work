#include "hip/hip_runtime.h"
#include "TimeIntegration.h"

#include <cusp\monitor.h>
#include <cusp\linear_operator.h>
#include <cusp\csr_matrix.h>
#include <cusp\krylov\bicgstab.h>
#include <cusp\gallery\poisson.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipsparse.h>
#include <thrust\device_ptr.h>
#include <hip/hip_runtime.h>

using namespace std;

static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
	switch (error)
	{

	case HIPSPARSE_STATUS_SUCCESS:
		return "HIPSPARSE_STATUS_SUCCESS";

	case HIPSPARSE_STATUS_NOT_INITIALIZED:
		return "HIPSPARSE_STATUS_NOT_INITIALIZED";

	case HIPSPARSE_STATUS_ALLOC_FAILED:
		return "HIPSPARSE_STATUS_ALLOC_FAILED";

	case HIPSPARSE_STATUS_INVALID_VALUE:
		return "HIPSPARSE_STATUS_INVALID_VALUE";

	case HIPSPARSE_STATUS_ARCH_MISMATCH:
		return "HIPSPARSE_STATUS_ARCH_MISMATCH";

	case HIPSPARSE_STATUS_MAPPING_ERROR:
		return "HIPSPARSE_STATUS_MAPPING_ERROR";

	case HIPSPARSE_STATUS_EXECUTION_FAILED:
		return "HIPSPARSE_STATUS_EXECUTION_FAILED";

	case HIPSPARSE_STATUS_INTERNAL_ERROR:
		return "HIPSPARSE_STATUS_INTERNAL_ERROR";

	case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
		return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

	case HIPSPARSE_STATUS_ZERO_PIVOT:
		return "HIPSPARSE_STATUS_ZERO_PIVOT";
	}

	return "<unknown>";
}

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
	if (HIPSPARSE_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUSPARSE error in file '%s', line %Ndims\Nobjs %s\nerror %Ndims: %s\nterminating!\Nobjs", __FILE__, __LINE__, err, \
			_cusparseGetErrorEnum(err)); \
			hipDeviceReset(); assert(0); \
	}
}

extern "C" void cusparseSafeCall(hipsparseStatus_t err) { __cusparseSafeCall(err, __FILE__, __LINE__); }

void TimeIntegration(Particle3D *particles, ReadWrite FileControl, data_in *input_data, int nump)
{
	int z, cont_bcon = 0;
	int max_nei = nump;						// N�mero m�ximo de vizinhos
	FILE *outNei = fopen("outNei.txt", "a");
	FILE *outTimes = fopen("outTimes.txt", "w");

	// Sistema
	//SystemResolution *sys;
	// --- Inicializando cuSPARSE e criando descritor da matriz esparsa A
	hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));
	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

	int division = nump / 1024;
	int mod = nump - (division * 1024);
	int DIM = 2;

	// Array de part�culas e alocado no device
	Particle3D *particles_d = NULL;
	hipError_t err = hipSuccess;
	err = hipMalloc((void **)&particles_d, nump*sizeof(Particle3D));
	
	/********************************************************/// Informa��o sobre a part�cula mais a direita e tempo da simula��o (Crista da onda x andamento da simula��o)
	/*FILE *outPointsDamBreak;
	errno_t errorCode = fopen_s(&outPointsDamBreak, "outPointsMPS.txt", "w");
	double maximum = 0.0;
	int location = 0;*/
	/********************************************************/

	// Inicializando e setando os arrays de vizinhos para a itera��o 0
	/*neighbor nei(nump, max_nei);
	neighbor neiICCG(nump, max_nei);

	nei.set(particles, input_data->radius2, nump);
	int *nei_1D = new int[nump*max_nei];
	for (int a = 0; a < nump; a++){
		for (int b = 0; b < max_nei; b++)	{
			nei_1D[(a * nump) + b] = nei.neighbors[a][b];
		}
	}

	neiICCG.set(particles, input_data->radius_ICCG2, nump);
	int *neiICCG_1D = new int[nump*max_nei];;
	for (int a = 0; a < nump; a++){
		for (int b = 0; b < max_nei; b++)	{
			neiICCG_1D[(a * nump) + b] = neiICCG.neighbors[a][b];
		}
	}
	*/
	int *nei_d = NULL;
	int *neiICCG_d = NULL;

	err = hipMalloc((void **)&nei_d, nump*max_nei*sizeof(int));
	err = hipMalloc((void **)&neiICCG_d, nump*max_nei*sizeof(int));
	hipMemset(nei_d, 0, sizeof(int) * nump * max_nei);
	hipMemset(neiICCG_d, 0, sizeof(int) * nump * max_nei);
	/*err = hipMemcpy(nei_d, nei_1D, nump*max_nei*sizeof(int), hipMemcpyHostToDevice);
	err = hipMemcpy(neiICCG_d, neiICCG_1D, nump*max_nei*sizeof(int), hipMemcpyHostToDevice);*/
	
	// Array de part�culas transferido do host para o device
	err = hipMemcpy(particles_d, particles, nump*sizeof(Particle3D), hipMemcpyHostToDevice);
	//sys = new SystemResolution(particles, nump);

	// Inicializando e setando os arrays de vizinhos para a itera��o 0
	set_nei_kernel << <division, 1024 >> >(0, particles_d, input_data->radius2, nump, nei_d);
	set_nei_kernel << <1, mod >> >(division * 1024, particles_d, input_data->radius2, nump, nei_d);
	set_nei_kernel << <division, 1024 >> >(0, particles_d, input_data->radius_ICCG2, nump, neiICCG_d);
	set_nei_kernel << <1, mod >> >(division * 1024, particles_d, input_data->radius_ICCG2, nump, neiICCG_d);

	// Setando os valores de n (particle number density) de cada part�cula e tamb�m os valores de n0 & n0_iccg para a itera��o 0
	cal_n_kernel << <division, 1024 >> >(0, neiICCG_d, particles_d, input_data->radius_ICCG, nump);
	cal_n_kernel << <1, mod >> >(division * 1024, neiICCG_d, particles_d, input_data->radius_ICCG, nump);
	hipDeviceSynchronize();

	double n0pICCG = 0.0, *n0pICCG_d = NULL, *temp = new double[1];
	hipMalloc((void **)&n0pICCG_d, sizeof(double));
	set_n_kernel << <1, 1 >> >(particles_d, n0pICCG_d);
	hipDeviceSynchronize();
	hipMemcpy(temp, n0pICCG_d, sizeof(double), hipMemcpyDeviceToHost);
	hipFree(n0pICCG_d);
	n0pICCG = temp[0];
	cout << n0pICCG << endl;
	system("pause");

	cal_n_kernel << <division, 1024 >> >(0, nei_d, particles_d, input_data->radius, nump);
	cal_n_kernel << <1, mod >> >(division * 1024, nei_d, particles_d, input_data->radius, nump);
	hipDeviceSynchronize();

	double n0p = 0.0, *n0p_d = NULL;
	hipMalloc((void **)&n0p_d, sizeof(double));
	set_n_kernel << <1, 1 >> >(particles_d, n0p_d);
	hipDeviceSynchronize();
	hipMemcpy(temp, n0p_d, sizeof(double), hipMemcpyDeviceToHost);
	hipFree(n0p_d);
	n0p = temp[0];
	cout << n0p << endl;
	system("pause");

	// Setando valor de dt
	double *dt_d = NULL;
	temp[0] = input_data->dt;
	hipMalloc((void **)&dt_d, sizeof(double));
	hipMemcpy(dt_d, temp, sizeof(double), hipMemcpyHostToDevice);

	// Criando array pmin, temp_b, temp_a2, dpArray e DdvCorrector no DEVICE & bcon, contB, contTemp, srcB, temp_a, dp e srcA no HOST e no DEVICE
	double *pmin_d = NULL;
	hipMalloc((void **)&pmin_d, nump*sizeof(double));
	Point3D *DdvCorrector_d = NULL;
	hipMalloc((void **)&DdvCorrector_d, nump*sizeof(Point3D));
	double *temp_b_d = NULL;
	hipMalloc((void **)&temp_b_d, nump*sizeof(double));
	int *bcon = new int[nump];
	int *bcon_d = NULL;
	hipMalloc((void **)&bcon_d, nump*sizeof(int));
	int *bcon_cont = new int[nump];
	int *bcon_cont_d = NULL;
	hipMalloc((void **)&bcon_cont_d, nump*sizeof(int));
	int *contB = new int[1];
	int *contB_d = NULL;
	hipMalloc((void **)&contB_d, sizeof(int));
	hipMemset(contB_d, 0, sizeof(int));
	int *contTemp = new int[1];
	int *contTemp_d = NULL;
	hipMalloc((void **)&contTemp_d, sizeof(int));
	hipMemset(contTemp_d, 0, sizeof(int));
	double *srcB = new double[nump];
	double *srcB_d = NULL;
	hipMalloc((void **)&srcB_d, nump*sizeof(double));
	//double *temp_a = new double[nump*nump];
	double *temp_a_d = NULL;
	hipMalloc((void **)&temp_a_d, nump*nump*sizeof(double));
	double *temp_a2_d = NULL;
	hipMalloc((void **)&temp_a2_d, nump*nump*sizeof(double));
	double *srcA = new double[nump*nump];
	double *srcA_d = NULL;
	hipMalloc((void **)&srcA_d, nump*nump*sizeof(double));
	double *dp = new double[nump];
	double *dp_d = NULL;
	err = hipMalloc((void **)&dp_d, nump*sizeof(double));
	double *dpArray_d = NULL;
	err = hipMalloc((void **)&dpArray_d, nump*sizeof(double));

	/*size_t avail;
	size_t total;
	
	double used_db, free_db, total_db;

	hipEvent_t *t_begin, *t_end;
	int functionsNum = 20;
	t_begin = new hipEvent_t[functionsNum];
	t_end = new hipEvent_t[functionsNum];
	float *time_spent = new float[functionsNum];
	time_spent[19] = 0.0;

	for (int r = 0; r < functionsNum; r++) {
		hipEventCreate(&t_begin[r]);
		hipEventCreate(&t_end[r]);
	}*/
	
	for (int it = 1; it <= input_data->MaxIteration; it++)
	{
		//cout << it << endl;

		//hipEventRecord(t_begin[0]);
		// Calculando o valor de dt (passo de tempo)
		//set_cal_dt <<< 1, 1 >> > (0, particles_d, dt_d, input_data->DtRatio, input_data->AveParticleDis, input_data->MaxDt, nump);
		//hipEventRecord(t_end[0]);

		//hipEventRecord(t_begin[1]);
		// Calculando a influ�ncia das for�as externas nas part�cula, seus movimentos e colis�o entre elas 
		external_force_kernel << <division, 1024 >> >(0, particles_d, dt_d, input_data->Gravity);
		external_force_kernel << <1, mod >> >(division * 1024, particles_d, dt_d, input_data->Gravity);
		//hipEventRecord(t_end[1]);
	
		//hipEventRecord(t_begin[2]);
		mov_part_kernel << <division, 1024 >> >(0, particles_d, dt_d);
		mov_part_kernel << <1, mod >> >(division * 1024, particles_d, dt_d);
		//hipEventRecord(t_end[2]);

		//hipEventRecord(t_begin[3]);
		collision_kernel << <division, 1024 >> >(0, particles_d, dt_d, nei_d, input_data->radius2lim, input_data->DensityFluid0, nump);
		collision_kernel << <1, mod >> >(division * 1024, particles_d, dt_d, nei_d, input_data->radius2lim, input_data->DensityFluid0, nump);
		//hipEventRecord(t_end[3]);
		hipDeviceSynchronize();

		// Limpando e recalculando o array de vizinhos
		hipMemset(nei_d, 0, sizeof(int) * nump * max_nei);
		hipMemset(neiICCG_d, 0, sizeof(int) * nump * max_nei);

		//hipEventRecord(t_begin[4]);
		set_nei_kernel << <division, 1024 >> >(0, particles_d, input_data->radius2, nump, nei_d);
		set_nei_kernel << <1, mod >> >(division * 1024, particles_d, input_data->radius2, nump, nei_d);
		set_nei_kernel << <division, 1024 >> >(0, particles_d, input_data->radius_ICCG2, nump, neiICCG_d);
		set_nei_kernel << <1, mod >> >(division * 1024, particles_d, input_data->radius_ICCG2, nump, neiICCG_d);
		//hipEventRecord(t_end[4]);

		//hipEventRecord(t_begin[5]);
		// Recalculando n (particle number density)
		cal_n_kernel << <division, 1024 >> >(0, nei_d, particles_d, input_data->radius, nump);
		cal_n_kernel << <1, mod >> >(division * 1024, nei_d, particles_d, input_data->radius, nump);
		//hipEventRecord(t_end[5]);

		//hipEventRecord(t_begin[6]);
		// Setando condi��o de contorno
		set_bcon_kernel << <division, 1024 >> >(0, particles_d, n0p, input_data->dirichlet, nump, bcon_d);
		set_bcon_kernel << <1, mod >> >(division * 1024, particles_d, n0p, input_data->dirichlet, nump, bcon_d);
		//hipEventRecord(t_end[6]);
		hipDeviceSynchronize();
		
		// Vetor B (source term)
		hipMemset(temp_b_d, 0, sizeof(double) * nump);
		hipMemset(contTemp_d, 0, sizeof(int));

		//hipEventRecord(t_begin[7]);
		cal_B_HS_ECS_kernel << <division, 1024 >> >(0, particles_d, n0p, dt_d, nei_d, input_data->radius, nump, bcon_d, contTemp_d, temp_b_d);
		cal_B_HS_ECS_kernel << <1, mod >> >(division * 1024, particles_d, n0p, dt_d, nei_d, input_data->radius, nump, bcon_d, contTemp_d, temp_b_d);
		//hipEventRecord(t_end[7]);
		hipDeviceSynchronize();

		/*cal_B_kernel << <division, 1024 >> >(0, particles_d, n0p, dt_d, contTemp_d, temp_b_d);
		cal_B_kernel << <1, mod >> >(division * 1024, particles_d, n0p, dt_d, contTemp_d, temp_b_d);
		hipDeviceSynchronize();*/


		hipMemset(srcB_d, 0, sizeof(double) * nump);
		hipMemset(contB_d, 0, sizeof(int));

		//hipEventRecord(t_begin[8]);
		set_B_HS_ECS_kernel << <1, 1 >> >(0, bcon_d, contB_d, srcB_d, temp_b_d, contTemp_d);
		//hipEventRecord(t_end[8]);
		hipDeviceSynchronize();

		// Matriz A (Laplaciano)
		hipMemset(temp_a_d, 0, sizeof(double) * nump*nump);
		//hipEventRecord(t_begin[9]);
		cal_A_HL_kernel << <division, 1024 >> >(0, particles_d, neiICCG_d, n0pICCG, input_data->radius_ICCG, input_data->DensityFluid0, DIM, nump, bcon_d, temp_a_d);
		cal_A_HL_kernel << <1, mod >> >(division * 1024, particles_d, neiICCG_d, n0pICCG, input_data->radius_ICCG, input_data->DensityFluid0, DIM, nump, bcon_d, temp_a_d);
		//hipEventRecord(t_end[9]);
		hipDeviceSynchronize();

		/*err = hipMemcpy(temp_a, temp_a_d, nump*nump*sizeof(double), hipMemcpyDeviceToHost);
		for (int g = 0; g < nump*nump; g++){
			if (g > 0 && g%nump == 0) fprintf(outNei, "\n");
			fprintf(outNei,"%lf ", temp_a[g]);
		}*/

		err = hipMemcpy(contB, contB_d, sizeof(int), hipMemcpyDeviceToHost);
		err = hipMemcpy(bcon, bcon_d, nump*sizeof(int), hipMemcpyDeviceToHost);

		cont_bcon = 0;
		for (int i = 0; i < nump; i++){
			if (bcon[i] != 0){
				bcon_cont[i] = cont_bcon;
				cont_bcon++;
			}
			else{
				bcon_cont[i] = cont_bcon;
			}
		}
		
		hipMemset(srcA_d, 0, sizeof(double) * nump*nump);
		err = hipMemcpy(bcon_cont_d, bcon_cont, nump*sizeof(int), hipMemcpyHostToDevice);

		//hipEventRecord(t_begin[10]);
		set_A_HL_kernel1 << <division, 1024 >> >(0, bcon_d, temp_a2_d, temp_a_d, nump, bcon_cont_d);
		set_A_HL_kernel1 << <1, mod >> >(division * 1024, bcon_d, temp_a2_d, temp_a_d, nump, bcon_cont_d);
		//hipEventRecord(t_end[10]);
		hipDeviceSynchronize();

		/*err = hipMemcpy(temp_a, temp_a_d, nump*nump*sizeof(double), hipMemcpyDeviceToHost);
		for (int g = 0; g < nump*nump; g++){
			if (g > 0 && g%nump == 0) fprintf(outNei, "\n");
			fprintf(outNei, "%lf ", temp_a[g]);
		}*/

		if ((contB[0]/1024) < 1){ 
			//hipEventRecord(t_begin[11]);
			set_A_HL_kernel2 << <contB[0], contB[0] >> >(0, 0, srcA_d, contB_d, temp_a2_d, nump);
			//hipEventRecord(t_end[11]);
		}
		else{			
			//hipEventRecord(t_begin[11]);
			//for (int y = 0; y < contB[0] / 1024; y++){
				for (z = 0; z < contB[0] / 1024; z++){
					//set_A_HL_kernel2 << < 1024, 1024 >> >(y, z, srcA_d, contB_d, temp_a2_d, nump);
					set_A_HL_kernel2 << < contB[0], 1024 >> >(0, z, srcA_d, contB_d, temp_a2_d, nump);
					hipDeviceSynchronize();
				}
			//}
			set_A_HL_kernel2 << <contB[0], (contB[0] % 1024) >> >(0, z, srcA_d, contB_d, temp_a2_d, nump);
			//hipEventRecord(t_end[11]);
			
			//for (int it = 0; it < contB[0] / 1024; it++){
				//set_A_HL_kernel2 << <1024, (contB[0] % 1024) >> >(it, z + 1, srcA_d, contB_d, temp_a2_d, nump);
				//set_A_HL_kernel2 << <(contB[0] % 1024), 1024 >> >(y + 1, it, srcA_d, contB_d, temp_a2_d, nump);
			//}			
			//set_A_HL_kernel2 << <(contB[0] % 1024), (contB[0] % 1024) >> >(y+1, z+1, srcA_d, contB_d, temp_a2_d, nump);
		}
		hipDeviceSynchronize();


		/*for (int g = 0; g < nump*nump; g++){
			if (g > 0 && g%nump == 0) fprintf(outNei, "\n");
			fprintf(outNei,"%lf ", srcA[g]);
		}*/

		// Resolu��o do sistema linear

		/*if (nump > 10000){
			hipMemcpy(srcA, srcA_d, nump*nump*sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(srcB, srcB_d, nump*sizeof(double), hipMemcpyDeviceToHost);
			sys->system_resolution(input_data->MaxIterPress, input_data->MinIterPress, input_data->Convergence, nump, bcon, contB[0], srcB, srcA);
			dp = sys->get_dp();
			hipMemcpy(dp_d, dp, nump*sizeof(double), hipMemcpyHostToDevice);
		}
		else{
*/
			/*************************************************************************************************************************************************/
			int nnz = 0;                                // Onde � guardado o n�mero de elementos diferentes de zero na matriz A em sua forma densa
			const int lda = contB[0];                   // Dimens�o principal da matriz A
			// Descobrindo o n�mero de elementos diferentes de zero e o n.e.d.z. por linha na matriz A (em sua forma densa)
			int *d_nnzPerVector;
			hipMalloc(&d_nnzPerVector, contB[0] * sizeof(*d_nnzPerVector));
			/*cusparseSafeCall*/(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, contB[0], contB[0], descrA, srcA_d, lda, d_nnzPerVector, &nnz));

			// Arrays para armazenar a matriz A no formato CSR (compressed sparse row)
			double *d_A;            (hipMalloc(&d_A, nnz * sizeof(*d_A)));
			int *d_A_RowIndices;    (hipMalloc(&d_A_RowIndices, (contB[0] + 1) * sizeof(*d_A_RowIndices)));
			int *d_A_ColIndices;    (hipMalloc(&d_A_ColIndices, nnz * sizeof(*d_A_ColIndices)));

			// Ponteiros para o device indicando o in�cio de cada um dos array
			thrust::device_ptr<double> dev_ptr_d_A = thrust::device_pointer_cast(d_A);
			thrust::device_ptr<int> dev_ptr_d_A_RowIndices = thrust::device_pointer_cast(d_A_RowIndices);
			thrust::device_ptr<int> dev_ptr_d_A_ColIndices = thrust::device_pointer_cast(d_A_ColIndices);
			thrust::device_ptr<double> dev_ptr_srcB_d = thrust::device_pointer_cast(srcB_d);
			//thrust::device_ptr<int> dev_ptr_dpArray = thrust::device_pointer_cast(dpArray_d);

			// Convertendo de fato a matrix densa em CSR
			/*cusparseSafeCall*/(hipsparseDdense2csr(handle, contB[0], contB[0], descrA, srcA_d, lda, d_nnzPerVector, d_A, d_A_RowIndices, d_A_ColIndices));

			cusp::csr_matrix<int, double, cusp::device_memory> csrA;
			csrA.resize(contB[0], contB[0], nnz);
			csrA.row_offsets.assign(dev_ptr_d_A_RowIndices, dev_ptr_d_A_RowIndices + contB[0] + 1);
			csrA.column_indices.assign(dev_ptr_d_A_ColIndices, dev_ptr_d_A_ColIndices + nnz);
			csrA.values.assign(dev_ptr_d_A, dev_ptr_d_A + nnz);

			cusp::array1d<double, cusp::device_memory> x(csrA.num_rows, 0);
			cusp::array1d<double, cusp::device_memory> array1dB(dev_ptr_srcB_d, dev_ptr_srcB_d + contB[0]);

			// Configurando crit�rio de parada:
			//  Limite de itera��es = 100
			//  Toler�ncia relativa = 1e-16
			//  Toler�ncia absoluta = 0
			//  Verbose            = true
			cusp::monitor<double> monitor(array1dB, 100, 1e-16, 0, false);

			// Configurando precondicionador (identidade)
			cusp::identity_operator<double, cusp::device_memory> M(csrA.num_rows, csrA.num_rows);

			// Resolvendo o sistema linear A x = b
			//hipEventRecord(t_begin[12]);
			cusp::krylov::bicgstab(csrA, x, array1dB, monitor, M);
			//hipEventRecord(t_end[12]);

			// Armazenando o resultado (press�es das part�culas)
			//cusp::array1d<double, cusp::host_memory> array1d_dp(x);
			//for (int i = 0; i < contB[0]; i++) dp[i] = array1d_dp[i];
			dpArray_d = thrust::raw_pointer_cast(&x[0]);

			// Desalocando os arrays e matriz do device
			{
				cusp::csr_matrix<int, double, cusp::device_memory> tmp;
				csrA.swap(tmp);
				cusp::array1d<double, cusp::device_memory> temp;
				x.swap(temp);
				array1dB.swap(temp);
			}
			hipFree(d_nnzPerVector);
			hipFree(d_A);
			hipFree(d_A_ColIndices);
			hipFree(d_A_RowIndices);

			/*************************************************************************************************************************************************/

			// Setando press�es das part�culas 
			//err = hipMemcpy(dpArray_d, dp, nump*sizeof(double), hipMemcpyHostToDevice);

			//hipEventRecord(t_begin[13]);
			conf_dp_kernel << <1, 1 >> >(dp_d, dpArray_d, nump, bcon_d, contB_d);
			//hipEventRecord(t_end[13]);
		//}

		//hipEventRecord(t_begin[14]);
		set_pr_kernel << <division, 1024>> >(0, particles_d, dp_d);
		set_pr_kernel << <1, mod >> >(division * 1024, particles_d, dp_d);
		//hipEventRecord(t_end[14]);

		//hipEventRecord(t_begin[15]);
		// C�lculo do gradiente da velocidade -> atualiza��o das velocidades
		set_pmin_kernel << <division, 1024 >> >(0,  particles_d, neiICCG_d, nump, pmin_d);
		set_pmin_kernel << <1, mod >> >(division * 1024, particles_d, neiICCG_d, nump, pmin_d);
		//hipEventRecord(t_end[15]);

		//hipEventRecord(t_begin[16]);
		set_ddv_kernel << <division, 1024 >> >(0, particles_d, neiICCG_d, input_data->DensityFluid0, dt_d, n0p, input_data->radius, DIM, nump, pmin_d, DdvCorrector_d);
		set_ddv_kernel << <1, mod >> >(division * 1024, particles_d, neiICCG_d, input_data->DensityFluid0, dt_d, n0p, input_data->radius, DIM, nump, pmin_d, DdvCorrector_d);
		//hipEventRecord(t_end[16]);

		////hipEventRecord(t_begin[17]);
		updateVelocity_kernel << <division, 1024 >> >(0, particles_d, dt_d, nump, DdvCorrector_d);
		updateVelocity_kernel << <1, mod >> >(division * 1024, particles_d, dt_d, nump, DdvCorrector_d);
		//hipEventRecord(t_end[17]);
		hipDeviceSynchronize();

		/*time_spent[18] = 0.0;
		for (int r = 0; r <= 17; r++) {
			hipEventElapsedTime(&time_spent[r], t_begin[r], t_end[r]);
			fprintf(outTimes, "Fun��o #%d: %lf\n", r, time_spent[r]);
			time_spent[18] = time_spent[18] + time_spent[r];
		}
		fprintf(outTimes, "Tempo total decorrido devido �s fun��es CUDA em milissegundos: %lf\n", time_spent[18]);
		time_spent[19] = time_spent[19] + time_spent[18];*/

		// Movendo informa��es das part�culas do device para o host para impress�o nos VTUs
		err = hipMemcpy(particles, particles_d, nump*sizeof(Particle3D), hipMemcpyDeviceToHost);

		// Escrevendo nos VTUs a configua��o do sistema no time step atual
		FileControl.WriteOut(particles, it, nump);

		// Informa��o sobre a part�cula mais a direita e tempo da simula��o (Crista da onda x andamento da simula��o)
		/*************************************************************************************///ALBVS begin
		//for (int k = 1; k < nump; k++){
		//	if ((particles[k].get_po().x > maximum) && (abs(particles[k].get_v().x) + abs(particles[k].get_v().y) != 0.0)){
		//		maximum = particles[k].get_po().x;
		//		location = k + 1;
		//	}
		//}
		//fprintf(outPointsDamBreak, "%lf %lf\n"/*, (maximum - 0.02)*/, ((it-1)*0.00025)*sqrt(9.8 / 0.6), (maximum - 0.02) / 0.6); // L = 0.6 m, g = 9.8 m/s�, t = it*10^-3 s
		/************************************************************************************///ALBVS end
		//sys->clear(nump);
	}
	//fprintf(outTimes, "Tempo total da simula��o devido �s fun��es CUDA em milissegundos: %lf\n", time_spent[19]);

	/*hipMemGetInfo(&avail, &total);
	free_db = (double)avail;

	total_db = (double)total;

	used_db = total_db - free_db;

	printf("GPU memory usage: used = %lf MB, free = %lf MB, total = %lf MB\n",

	used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);

	system("pause");*/

	hipFree(pmin_d);
	hipFree(bcon_d);
	hipFree(bcon_cont_d);
	hipFree(dp_d);
	hipFree(dpArray_d);
	hipFree(particles_d);
	hipFree(dt_d);
	hipFree(bcon_d);
	hipFree(nei_d);
	hipFree(neiICCG_d);
	hipFree(srcB_d);
	hipFree(temp_b_d);
	hipFree(contB_d);
	hipFree(contTemp_d);
	hipFree(srcA_d);
	hipFree(temp_a_d);
	hipFree(temp_a2_d);
	delete[] dp;
	/*delete[] nei_1D;
	delete[] neiICCG_1D;*/
	delete[] bcon;
	delete[] bcon_cont;
	delete[] srcB;
	//delete[] temp_a;
	delete[] contB;
	delete[] contTemp;
	delete[] srcA;
	fclose(outNei);
	fclose(outTimes);
	//fclose(outPointsDamBreak);
}