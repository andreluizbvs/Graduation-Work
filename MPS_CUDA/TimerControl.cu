#include "hip/hip_runtime.h"
#include "TimerControl.h"

// Controle de tempo de cada intera��o: O tempo varia de acordo com a fun��o abaixo

__global__ void set_cal_dt(int offset, Particle3D* particles, double *dt, double DtRatio, double AveParticleDis, double MaxDt, int nump)
{
	double vmax = 0;
	double v = 0;
	double dt_limt = 0;
	//double dt = 0;

	for (int a = 0; a<nump; a++)
	{
		if ((particles)[a].is_fluid())
		{
			v = pow((particles)[a].get_v().x, 2);
			v += pow((particles)[a].get_v().y, 2);
			v += pow((particles)[a].get_v().z, 2);

			if (v>vmax)
				vmax = v;
		}
	}

	vmax = sqrt(vmax);

	dt_limt = dt[0]*INCREASE_DT;

	if (vmax == 0.0){
		dt[0] = dt_limt;
	}else{

		dt[0] = DtRatio*AveParticleDis / vmax;

		if (dt[0] > dt_limt)
			dt[0] = dt_limt;

		if (dt[0] > MaxDt)
			dt[0] = MaxDt;
	}
}

double TimerControl::cal_dt(Particle2D* particles, double OldDt, double DtRatio, double AveParticleDis, double MaxDt, int nump)
{
	double vmax = 0;
	double v = 0;
	double dt_limt = 0;
	double dt = 0;

	for (int a = 0; a<nump; a++)
	{
		if ((particles)[a].is_fluid())
		{
			v = pow((particles)[a].get_v().x, 2);
			v += pow((particles)[a].get_v().y, 2);

			if (v>vmax)
				vmax = v;
		}
	}

	vmax = sqrt(vmax);

	dt_limt = OldDt*INCREASE_DT;

	if (vmax == 0.0)
		return dt_limt;

	dt = DtRatio*AveParticleDis / vmax;

	if (dt>dt_limt)
		dt = dt_limt;

	if (dt>MaxDt)
		dt = MaxDt;

	return dt;
}

TimerControl::TimerControl()
{

}

TimerControl::~TimerControl()
{

}

TimerControl::TimerControl(const TimerControl &t)
{

}