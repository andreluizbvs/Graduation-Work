#include "hip/hip_runtime.h"
#include "density.h"

__global__ void set_n_kernel(Particle3D* particles, double* n0p)
{
	n0p[0] = particles[0].get_n();
}

__global__ void cal_n_kernel(int offset, int* nei_1D, Particle3D* particles, double r, int nump)
{
	unsigned int i = offset + (blockDim.x * blockIdx.x + threadIdx.x);

	int j = 0;
	double dx = 0.0, dy = 0.0, dz = 0.0, dist = 0.0, kern = 0.0, n = 0.0;
	Point3D Part_j, Part_i;
	kernel k;

	n = 0.0;

	for (int a = 1; a <= nei_1D[(i*nump) + 0]; a++)
	{
		j = nei_1D[(i *nump) + a];

		Part_j = particles[j].get_po();
		Part_i = particles[i].get_po();

		dx = Part_j.x - Part_i.x;
		dy = Part_j.y - Part_i.y;
		dz = Part_j.z - Part_i.z;

		dist = sqrt(dx*dx + dy*dy + dz*dz);

		kern = k.weight(dist, r);

		n += kern;
	}
	particles[i].set_n(n);
}

__host__ __device__ void density::cal_n2D(neighbor nei, Particle2D* particles, double r, int nump)
{
	int j = 0;
	double dx = 0.0, dy = 0.0, dist = 0.0, k = 0.0, n = 0.0;
	Point2D Part_j, Part_i;
	for (int i = 0; i < nump; i++)
	{
		n = 0.0;

		for (int a = 1; a <= nei.get()[i][0]; a++)
		{
			j = nei.get()[i][a];

			Part_j = particles[j].get_po();
			Part_i = particles[i].get_po();

			dx = Part_j.x - Part_i.x;
			dy = Part_j.y - Part_i.y;

			dist = sqrt(dx*dx + dy*dy);

			k = this->k.weight(dist, r);

			n += k;
		}
		particles[i].set_n(n);
	}
}

__host__ __device__ density::density()
{

}

__host__ __device__ density::~density()
{

}

__host__ __device__ density::density(const density &d)
{
	this->k = d.k;
}