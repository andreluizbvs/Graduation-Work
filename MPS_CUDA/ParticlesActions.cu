#include "hip/hip_runtime.h"
#include "ParticlesActions.h"

__global__ void set_pr_kernel(int offset, Particle3D* particles, double *dp)
{
	unsigned int a = offset + (blockDim.x * blockIdx.x + threadIdx.x);

	particles[a].set_pr(dp[a]);
}

__global__ void external_force_kernel(int offset, Particle3D* particles, double *dt, double g)
{
	unsigned int a = offset + (blockDim.x * blockIdx.x + threadIdx.x);

	if (particles[a].is_fluid()){
		particles[a].set_v(particles[a].get_v().x, particles[a].get_v().y - g*dt[0], (particles[a].get_v().z));
	}
}

__global__ void mov_part_kernel(int offset, Particle3D* particles, double *dt)
{
	unsigned int a = offset + (blockDim.x * blockIdx.x + threadIdx.x);

	if ((particles)[a].is_fluid()){
		(particles)[a].set_po((particles)[a].get_po().x + (particles)[a].get_v().x*dt[0], (particles)[a].get_po().y + (particles)[a].get_v().y*dt[0], (particles)[a].get_po().z + (particles)[a].get_v().z*dt[0]);
	}
}

__global__ void collision_kernel(int offset, Particle3D* particles, double *dt, int *nei_1D, double radius, double Density, int nump)
{
	unsigned int i = offset + (blockDim.x * blockIdx.x + threadIdx.x);

	double dx = 0.0, dy = 0.0, dz = 0.0, dist = 0.0;
	double vg[3], vr[3], vbas, vm[3];
	double m1 = 0.0, m2 = 0.0;
	Point3D Part_j, Part_i, Velocity_j, Velocity_i;

	int j = 0;;

	m1 = Density;

	for (int a = 1; a <= nei_1D[(i*nump) + 0]; a++)
	{
		j = nei_1D[(i*nump) + a];

		if (j <= i)
			continue;

		Part_j = (particles)[j].get_po();
		Part_i = (particles)[i].get_po();

		dx = Part_j.x - Part_i.x;
		dy = Part_j.y - Part_i.y;
		dz = Part_j.z - Part_i.z;

		dist = dx*dx + dy*dy + dz*dz;
		if (dist<radius)
		{
			Velocity_j = (particles)[j].get_v();
			Velocity_i = (particles)[i].get_v();

			dist = sqrt(dist);

			m2 = Density;
			vg[0] = (m1*Velocity_i.x + m2*Velocity_j.x) / (m1 + m2);
			vg[1] = (m1*Velocity_i.y + m2*Velocity_j.y) / (m1 + m2);
			vg[2] = (m1*Velocity_i.z + m2*Velocity_j.z) / (m1 + m2);

			vr[0] = m1*(Velocity_i.x - vg[0]);
			vr[1] = m1*(Velocity_i.y - vg[1]);
			vr[2] = m1*(Velocity_i.z - vg[2]);

			vbas = (vr[0] * dx + vr[1] * dy + vr[2] * dz) / dist;

			if (vbas<0.0)continue;

			vm[0] = (1.2)*vbas*dx / dist;
			vm[1] = (1.2)*vbas*dy / dist;
			vm[2] = (1.2)*vbas*dz / dist;

			if ((particles)[i].is_fluid())
			{
				(particles)[i].set_v(Velocity_i.x - vm[0] / m1, Velocity_i.y - vm[1] / m1, Velocity_i.z - vm[2] / m1);
				(particles)[i].set_po(Part_i.x - dt[0] * vm[0] / m1, Part_i.y - dt[0] * vm[1] / m1, Part_i.z - dt[0] * vm[2] / m1);
			}

			if ((particles)[j].is_fluid())
			{
				(particles)[j].set_v(Velocity_j.x + vm[0] / m2, Velocity_j.y + vm[1] / m2, Velocity_j.z + vm[2] / m2);
				(particles)[j].set_po(Part_j.x + dt[0] * vm[0] / m2, Part_j.y + dt[0] * vm[1] / m2, Part_j.z + dt[0] * vm[2] / m2);
			}
		}
	}
}

//void ParticlesActions::external_force_cu(Particle2D* particles, double dt, double g, int nump){
//
//	Particle2D *particles_d = NULL;
//	hipError_t err = hipSuccess;
//	err = hipMalloc((void **)&particles_d, nump*sizeof(Particle2D));
//	
//	if (err != hipSuccess)
//	{
//		fprintf(stderr, "Failed to allocate device particles (error code %s)!\n", hipGetErrorString(err));
//		exit(EXIT_FAILURE);
//	}
//	
//	int division = nump / 1024;
//	int mod = nump - (division * 1024);
//	err = hipMemcpy(particles_d, particles, nump*sizeof(Particle2D), hipMemcpyHostToDevice);
//	
//	if (err != hipSuccess)
//	{
//		fprintf(stderr, "Failed to copy particles from host to device (error code %s)!\n", hipGetErrorString(err));
//		exit(EXIT_FAILURE);
//	}
//	
//	external_force_kernel << <division, 1024 >> >(0, particles_d, dt, g);
//	external_force_kernel << <1, mod >> >(division * 1024, particles_d, dt, g);
//	
//	err = hipGetLastError();
//	if (err != hipSuccess)
//	{
//		fprintf(stderr, "Failed to launch external force kernel (error code %s)!\n", hipGetErrorString(err));
//		system("pause");
//		exit(EXIT_FAILURE);
//	}
//	
//	err = hipMemcpy(particles, particles_d, nump*sizeof(Particle2D), hipMemcpyDeviceToHost);
//	if (err != hipSuccess)
//	{
//		fprintf(stderr, "Failed to copy particles from device to host (error code %s)!\n", hipGetErrorString(err));
//		exit(EXIT_FAILURE);
//	}
//	
//	err = hipFree(particles_d);
//
//	if (err != hipSuccess)
//	{
//		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
//		exit(EXIT_FAILURE);
//	}
//	
//	err = hipDeviceReset();
//
//	if (err != hipSuccess)
//	{
//		fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
//		exit(EXIT_FAILURE);
//	}
//
//	//return particles;
//}