#include "hip/hip_runtime.h"
#include "velocityCorrection.h"

// Corre��o das velocidades e das posi��es das part�culas de acordo com os valores de press�o calculados.

__global__ void set_pmin_kernel(int offset, Particle3D* particles, int *nei, int nump, double *pmin){
	unsigned int i = offset + (blockDim.x * blockIdx.x + threadIdx.x);
	
	//double *pmin = new double[nump];
	double AuxPmin = 0.0;
	int j = 0;
	double p;

	if (particles[i].is_fluid()){
		AuxPmin = (particles)[i].get_pr();
		for (int a = 1; a <= nei[(i*nump) + 0]; a++){
			j = nei[(i*nump) + a];

			if ((particles)[j].is_wall()){
			}
			else{
				p = (particles)[j].get_pr();

				if (p < AuxPmin)
					AuxPmin = p;
			}
		}
		pmin[i] = (AuxPmin);		
	}
	else{
		pmin[i] = 0.0;
	}
}

__global__ void set_ddv_kernel(int offset, Particle3D* particles, int *nei, double rho, double *dt, double n0p, double radius, double num_D, int nump, double *pmin, Point3D *DdvCorrector){
	unsigned int i = offset + (blockDim.x * blockIdx.x + threadIdx.x);

	Point3D Part_j, Part_i, new_point;
	int j = 0;
	double dx = 0.0, dy = 0.0, dz = 0.0, dist = 0.0, ddv = 0.0;
	//Point2D *DdvCorrector = new Point2D[nump];
	kernel k;

	new_point.x = 0.0;
	new_point.y = 0.0;
	new_point.z = 0.0;

	DdvCorrector[i] = (new_point);

	if ((particles)[i].is_fluid()){
		for (int a = 1; a <= nei[(i*nump) + 0]; a++){
			j = nei[(i*nump) + a];

			if (!particles[j].is_wall()){
				dx = 0.0;
				dy = 0.0;
				dz = 0.0;
				dist = 0.0;
				ddv = 0.0;

				Part_j = (particles)[j].get_po();
				Part_i = (particles)[i].get_po();

				dx = Part_i.x - Part_j.x;
				dy = Part_i.y - Part_j.y;
				dz = Part_i.z - Part_j.z;

				dist = sqrt(dx*dx + dy*dy + dz*dz);

				ddv = dt[0];

				ddv = ddv*(((particles)[i].get_pr() + (particles)[j].get_pr()) - (pmin[i] + pmin[j]));

				ddv = ddv / dist *k.weight(dist, radius);
				ddv = ddv / rho;
				ddv = ddv / n0p;
				ddv = ddv* num_D;

				DdvCorrector[i].x += ((ddv * dx) / dist);
				DdvCorrector[i].y += ((ddv * dy) / dist);
				DdvCorrector[i].z += ((ddv * dz) / dist);
			}
		}
	}
}

__global__ void updateVelocity_kernel(int offset, Particle3D* particles, double* dt, int nump, Point3D *DdvCorrector){
	unsigned int i = offset + (blockDim.x * blockIdx.x + threadIdx.x);
	
	Point3D v_i, Part_i;

	Part_i = (particles)[i].get_po();
	v_i = (particles)[i].get_v();

	if ((particles)[i].is_fluid())
	{
		(particles)[i].set_po(Part_i.x + DdvCorrector[i].x*dt[0], Part_i.y + DdvCorrector[i].y*dt[0], Part_i.z + DdvCorrector[i].z*dt[0]);
		(particles)[i].set_v(v_i.x + DdvCorrector[i].x, v_i.y + DdvCorrector[i].y, v_i.z + DdvCorrector[i].z);
	}
}

velocityCorrection::velocityCorrection()
{

}

velocityCorrection::~velocityCorrection()
{

}

velocityCorrection::velocityCorrection(velocityCorrection &v)
{
	//this->pmin = v.pmin;
	this->k = k;
}

void velocityCorrection::UpdateVelocity(Particle2D* particles, neighbor nei, double rho, double dt, double n0p, double radius, double num_D, int nump)
{
	double *pmin = new double[nump];
	double AuxPmin = 0.0;
	int j = 0;

	Point2D* DdvCorrector = new Point2D[nump];

	for (int i = 0; i < nump; i++)
	{
		if (!(particles)[i].is_fluid()){
			pmin[i] = 0.0;
			continue;
		}

		AuxPmin = (particles)[i].get_pr();
		for (int a = 1; a <= nei.get()[i][0]; a++)
		{
			j = nei.get()[i][a];

			if ((particles)[j].is_wall())
				continue;

			double p = (particles)[j].get_pr();

			if (p < AuxPmin)
				AuxPmin = p;
		}
		pmin[i] = (AuxPmin);
	}

	Point2D Part_j, Part_i;
	Point2D new_point;
	double dx = 0.0, dy = 0.0, dist = 0.0,/* kernel = 0.0,*/ ddv = 0.0;

	for (int i = 0; i < nump; i++)
	{
		new_point.x = 0.0;
		new_point.y = 0.0;

		DdvCorrector[i] = (new_point);

		if (!(particles)[i].is_fluid())
			continue;

		for (int a = 1; a <= nei.get()[i][0]; a++)
		{
			j = nei.get()[i][a];

			if ((particles)[j].is_wall())
				continue;

			dx = 0.0;
			dy = 0.0;
			dist = 0.0;
			//kernel = 0.0;
			ddv = 0.0;

			Part_j = (particles)[j].get_po();
			Part_i = (particles)[i].get_po();

			dx = Part_i.x - Part_j.x;
			dy = Part_i.y - Part_j.y;

			dist = sqrt(dx*dx + dy*dy);

			ddv = dt;

			ddv = ddv*(((particles)[i].get_pr() + (particles)[j].get_pr()) - (pmin[i] + pmin[j]));

			ddv = ddv / dist *k.weight(dist, radius);
			ddv = ddv / rho;
			ddv = ddv / n0p;
			ddv = ddv* num_D;

			DdvCorrector[i].x += ((ddv * dx) / dist);
			DdvCorrector[i].y += ((ddv * dy) / dist);
		}
	}

	Point2D v_i;

	for (int i = 0; i < nump; i++)
	{
		Part_i = (particles)[i].get_po();
		v_i = (particles)[i].get_v();

		if ((particles)[i].is_fluid())
		{
			(particles)[i].set_po(Part_i.x + DdvCorrector[i].x*dt, Part_i.y + DdvCorrector[i].y*dt);
			(particles)[i].set_v(v_i.x + DdvCorrector[i].x, v_i.y + DdvCorrector[i].y);
		}
	}
	delete[] pmin;
	delete[] DdvCorrector;
}
