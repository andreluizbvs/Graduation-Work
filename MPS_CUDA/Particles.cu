#include "Particles.h"

// Estrutura de uma part�cula 
// Uma particula tem posi��o, velocidade, press�o e densidade de n�mero de part�culas (densidade de vizinhan�a)

__host__ __device__ Particle2D::Particle2D()
{
	this->position.x = 0;
	this->position.y = 0;

	this->velocity.x = 0;
	this->velocity.y = 0;

	this->Type = 0;
	this->n = 0.0;
	this->p = 0.0;
}

__host__ __device__ Particle2D::~Particle2D()
{

}

__host__ __device__ Particle2D::Particle2D(const Particle2D &p)
{
	this->position = p.position;
	this->velocity = p.velocity;

	this->Type = p.Type;
	this->n = p.n;
	this->p = p.p;
}

__host__ __device__ void Particle2D::get_all(Point2D &position, Point2D &velocity, int &Type, double &n, double &p)
{
	position = this->position;
	velocity = this->velocity;

	Type = this->Type;
	n = this->n;
	p = this->p;
}

__host__ __device__ Point2D Particle2D::get_po()
{
	return this->position;
}

__host__ __device__ Point2D Particle2D::get_v()
{
	return this->velocity;
}

__host__ __device__ double  Particle2D::get_n()
{
	return this->n;
}

__host__ __device__ double Particle2D::get_pr()
{
	return this->p;
}

__host__ __device__ void Particle2D::set_all(Point2D position, Point2D velocity, int Type, double n, double p)
{
	this->position = position;
	this->velocity = velocity;

	this->Type = Type;

	this->n = n;
	this->p = p;
}

__host__ __device__ void Particle2D::set_n(double n)
{
	this->n = n;
}

__host__ __device__ void Particle2D::set_po(double x, double y)
{
	this->position.x = x;
	this->position.y = y;
}

__host__ __device__ void Particle2D::set_pr(double p)
{
	this->p = p;
}

__host__ __device__ void Particle2D::set_v(double x, double y)
{
	this->velocity.x = x;
	this->velocity.y = y;

}

__host__ __device__ bool Particle2D::is_border()
{
	if (Type == 2)
		return true;
	else
		return false;
}

__host__ __device__ bool Particle2D::is_fluid()
{
	if (Type == 0)
		return true;
	else
		return false;
}

__host__ __device__ bool Particle2D::is_wall()
{
	if (Type == 3)
		return true;
	else
		return false;
}

__host__ __device__ int Particle2D::get_type()
{
	return this->Type;
}


////////////////////////////////////////////////////////////////////////////////////////// 3 DIMENS�ES


__host__ __device__ Particle3D::Particle3D()
{
	this->position.x = 0;
	this->position.y = 0;
	this->position.z = 0;

	this->velocity.x = 0;
	this->velocity.y = 0;
	this->velocity.z = 0;

	this->Type = 0;
	this->n = 0.0;
	this->p = 0.0;
}

__host__ __device__ Particle3D::~Particle3D()
{

}

__host__ __device__ Particle3D::Particle3D(const Particle3D &p)
{
	this->position = p.position;
	this->velocity = p.velocity;

	this->Type = p.Type;
	this->n = p.n;
	this->p = p.p;
}

__host__ __device__ void Particle3D::get_all(Point3D &position, Point3D &velocity, int &Type, double &n, double &p)
{
	position = this->position;
	velocity = this->velocity;

	Type = this->Type;
	n = this->n;
	p = this->p;
}

__host__ __device__ Point3D Particle3D::get_po()
{
	return this->position;
}

__host__ __device__ Point3D Particle3D::get_v()
{
	return this->velocity;
}

__host__ __device__ double  Particle3D::get_n()
{
	return this->n;
}

__host__ __device__ double Particle3D::get_pr()
{
	return this->p;
}

__host__ __device__ void Particle3D::set_all(Point3D position, Point3D velocity, int Type, double n, double p)
{
	this->position = position;
	this->velocity = velocity;

	this->Type = Type;

	this->n = n;
	this->p = p;
}

__host__ __device__ void Particle3D::set_n(double n)
{
	this->n = n;
}

__host__ __device__ void Particle3D::set_po(double x, double y, double z)
{
	this->position.x = x;
	this->position.y = y;
	this->position.z = z;
}

__host__ __device__ void Particle3D::set_pr(double p)
{
	this->p = p;
}

__host__ __device__ void Particle3D::set_v(double x, double y, double z)
{
	this->velocity.x = x;
	this->velocity.y = y;
	this->velocity.z = z;

}

__host__ __device__ bool Particle3D::is_border()
{
	if (Type == 2)
		return true;
	else
		return false;
}

__host__ __device__ bool Particle3D::is_fluid()
{
	if (Type == 0)
		return true;
	else
		return false;
}

__host__ __device__ bool Particle3D::is_wall()
{
	if (Type == 3)
		return true;
	else
		return false;
}

__host__ __device__ int Particle3D::get_type()
{
	return this->Type;
}